// Includes
#include "../Geometry/geometry.h"
#include "../GeometricsObjects/forms.h"
#include "../Utils/camera.h"
#include "../Utils/scene.h"
#include <iostream>
#include <fstream>
#include <chrono>

using namespace std;
using namespace chrono;

/**
 * @param filename name of the image
 * @param image the image
 */
void savePPM(const string &filename, const Color *image)
{
    ofstream file(filename);
    if (!file.is_open())
        return;

    file << "P3\n"
         << WIDTH_PIXEL << " " << HEIGHT_PIXEL << "\n255\n";

    for (int y = 0; y < HEIGHT_PIXEL; y++)
    {
        for (int x = 0; x < WIDTH_PIXEL; x++)
        {
            const Color &color = image[y * WIDTH_PIXEL + x];
            file << color.r << " " << color.g << " " << color.b << " ";
        }
        file << "\n";
    }

    file.close();
}

int main(int argc, char *argv[])
{
    // -- Gestion des arguments (inchangée) --
    if (argc != 4)
    {
        fprintf(stderr, "Usage : %s [<nbsec> <fps> <nbturns>]\n", argv[0]);
        exit(1);
    }

    int nb_sec = atoi(argv[1]);
    int fps = atoi(argv[2]);
    int nb_turns = atoi(argv[3]);

    if (nb_sec > 300 || fps < 1 || nb_turns < 1)
    {
        fprintf(stderr, "Arguments invalides.\n");
        exit(2);
    }

    int nb_images = nb_sec * fps;
    system("rm -r ../build/video/*.ppm");
    system("rm ../build/output.mp4");
    char buffer[256];

    // -- Caméra et scène --
    Camera cam(Point3D(0, 0, 5), Vecteur3D(0, 0, -1), 90, WIDTH_PIXEL, HEIGHT_PIXEL, Vecteur3D(1, 0, 0), Vecteur3D(0, 1, 0));
    Scene scene(cam);

    // -- Lumières --
    scene.addLight(Light(Point3D(-1.5, 0, -5), Vecteur3D(0.5, 1, 0.5)));
    scene.addLight(Light(Point3D(1, 0, 5), Vecteur3D(0.4, 0.4, 1)));

    // -- Matériaux --
    Material matOrange(Vecteur3D(0.1, 0.1, 0.1), Vecteur3D(1, 0.307, 0.168), Vecteur3D(1, 1, 1), 300);

    // -- Objet : cube --
    Cube *cube = new Cube(3.0, Point3D(0, 0, 0), matOrange);
    scene.addTriangles(cube->triangles, 12);

    // -- Triangle device --
    Triangle *d_triangles;
    hipMalloc(&d_triangles, scene.numTriangles * sizeof(Triangle));
    hipMemcpy(d_triangles, scene.triangles, scene.numTriangles * sizeof(Triangle), hipMemcpyHostToDevice);

    // -- Lights device --
    Light *d_lights;
    hipMalloc(&d_lights, scene.numLights * sizeof(Light));
    hipMemcpy(d_lights, scene.lights, scene.numLights * sizeof(Light), hipMemcpyHostToDevice);

    // -- Camera device --
    Camera d_cam = {cam.position, cam.direction, cam.fov, cam.width, cam.height, Vecteur3D(1, 0, 0), Vecteur3D(0, 1, 0)};
    Camera *d_camera;
    hipMalloc(&d_camera, sizeof(Camera));
    hipMemcpy(d_camera, &d_cam, sizeof(Camera), hipMemcpyHostToDevice);

    // -- Image --
    Color *d_image;
    hipMalloc(&d_image, WIDTH_PIXEL * HEIGHT_PIXEL * sizeof(Color));

    auto t_start = high_resolution_clock::now();

    for (int i = 0; i < nb_images; i++)
    {
        sprintf(buffer, "video/frame%03d.ppm", i);

        // Rotation du cube
        cube->rotateX((180.0 * nb_turns) / nb_images, cube->getCenter());
        cube->rotateY((360.0 * nb_turns) / nb_images, cube->getCenter());

        // Copie des triangles mis à jour
        hipMemcpy(d_triangles, cube->triangles, scene.numTriangles * sizeof(Triangle), hipMemcpyHostToDevice);

        // Image host
        Color *h_image = new Color[WIDTH_PIXEL * HEIGHT_PIXEL];

        // Kernel
        dim3 blockDim(16, 16);
        dim3 gridDim((WIDTH_PIXEL + blockDim.x - 1) / blockDim.x, (HEIGHT_PIXEL + blockDim.y - 1) / blockDim.y);

        renderKernel<<<gridDim, blockDim>>>(d_image, d_triangles, scene.numTriangles, d_camera, d_lights, scene.numLights);

        hipDeviceSynchronize();

        // Copie vers host
        hipMemcpy(h_image, d_image, WIDTH_PIXEL * HEIGHT_PIXEL * sizeof(Color), hipMemcpyDeviceToHost);

        // Sauvegarde image
        savePPM(buffer, h_image);
        delete[] h_image;
    }

    auto t_end = high_resolution_clock::now();
    duration<double> t_total = t_end - t_start;

    // Nettoyage
    hipFree(d_image);
    hipFree(d_camera);
    hipFree(d_lights);
    hipFree(d_triangles);

    // Creation of the video using ffmpeg
    char ffmpegCommand[256];
    sprintf(ffmpegCommand, "ffmpeg -y -framerate %d -i ../build/video/frame%%03d.ppm -c:v libx264 -pix_fmt yuv420p output.mp4", fps);

    // Handle potential error
    int ret = system(ffmpegCommand);
    if (ret != 0)
    {
        cerr << "Erreur lors de la création de la vidéo avec ffmpeg." << endl;
    }

    cout << "Images générées avec FPS moyen : " << (nb_images / t_total.count()) << " (" << t_total.count() << "s)" << endl;

    return 0;
}
